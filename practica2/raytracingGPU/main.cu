#include "hip/hip_runtime.h"
//==================================================================================================
// Written in 2016 by Peter Shirley <ptrshrl@gmail.com>
//
// To the extent possible under law, the author(s) have dedicated all copyright and related and
// neighboring rights to this software to the public domain worldwide. This software is distributed
// without any warranty.
//
// You should have received a copy (see file COPYING.txt) of the CC0 Public Domain Dedication along
// with this software. If not, see <http://creativecommons.org/publicdomain/zero/1.0/>.
//==================================================================================================

#include <cstdio>
#include <cstdlib>

#include "raytracing.h"

#include "Vec3.h"
#include "Camera.h"
#include "Object.h"
#include "Scene.h"
#include "Sphere.h"
#include "Diffuse.h"
#include "Metallic.h"
#include "Crystalline.h"

#include "random.h"
#include "utils.h"

Scene randomScene() {
	Scene list;
	list.add(new Object(
		new Sphere(Vec3(0.0f, -1000.0f, 0.0f), 1000.0f),
		new Diffuse(Vec3(0.5f, 0.5f, 0.5f))
	));

	for (int a = -11; a < 11; a++) {
		for (int b = -11; b < 11; b++) {
			float choose_mat = random();
			Vec3 center(a + 0.9f * random(), 0.2f, b + 0.9f * random());
			if ((center - Vec3(4.0f, 0.2f, 0.0f)).length() > 0.9f) {
				if (choose_mat < 0.8f) {  // diffuse
					list.add(new Object(
						new Sphere(center, 0.2f),
						new Diffuse(Vec3(random() * random(),
							random() * random(),
							random() * random()))
					));
				} else if (choose_mat < 0.95f) { // metallic
					list.add(new Object(
						new Sphere(center, 0.2f),
						new Metallic(Vec3(0.5f * (1.0f + random()),
							0.5f * (1.0f + random()),
							0.5f * (1.0f + random())),
							0.5f * random())
					));
				} else {  // crystalline
					list.add(new Object(
						new Sphere(center, 0.2f),
						new Crystalline(1.5f)
					));
				}
			}
		}
	}

	list.add(new Object(
		new Sphere(Vec3(0.0f, 1.0f, 0.0f), 1.0f),
		new Crystalline(1.5f)
	));
	list.add(new Object(
		new Sphere(Vec3(-4.0f, 1.0f, 0.0f), 1.0f),
		new Diffuse(Vec3(0.4f, 0.2f, 0.1f))
	));
	list.add(new Object(
		new Sphere(Vec3(4.0f, 1.0f, 0.0f), 1.0f),
		new Metallic(Vec3(0.7f, 0.6f, 0.5f), 0.0f)
	));

	return list;
}

void rayTracingCPU(Vec3* img, int w, int h, int ns = 10) {
	Scene world = randomScene();
	world.setSkyColor(Vec3(0.5f, 0.7f, 1.0f));
	world.setInfColor(Vec3(1.0f, 1.0f, 1.0f));

	Vec3 lookfrom(13.0f, 2.0f, 3.0f);
	Vec3 lookat(0.0f, 0.0f, 0.0f);
	float dist_to_focus = 10.0f;
	float aperture = 0.1f;

	Camera cam(lookfrom, lookat, Vec3(0.0f, 1.0f, 0.0f), 20.0f, float(w) / float(h), aperture, dist_to_focus);

	for (int j = h - 1; j >= 0; j--) {
		for (int i = 0; i < w; i++) {
			Vec3 col(0.0f, 0.0f, 0.0f);
			for (int s = 0; s < ns; s++) {
				float u = float(i + random()) / float(w);
				float v = float(j + random()) / float(h);
				Ray r = cam.get_ray(u, v);
				col += world.getSceneColor(r);
			}
			col /= float(ns);
			col[0] = sqrt(col[0]);
			col[1] = sqrt(col[1]);
			col[2] = sqrt(col[2]);
			img[j * w + i] = col;
		}
	}
}

int main() {
	int w = 512;// 1200;
	int h = 256;// 800;
	int ns = 10;
	clock_t start, stop;
	double timer_seconds;

	size_t size = sizeof(unsigned char) * w * h * 3;
	unsigned char* data = (unsigned char*)malloc(size);

	Vec3* img;
	size_t isize = w * h * sizeof(Vec3);
	hipMallocManaged((void**)&img, isize);

	std::cerr << "--- CPU ---\n";
	start = clock();
	rayTracingCPU(img, w, h, ns);

	for (int i = h - 1; i >= 0; i--) {
		for (int j = 0; j < w; j++) {
			size_t idx = i * w + j;
			data[idx * 3 + 0] = char(255.99 * img[idx].b());
			data[idx * 3 + 1] = char(255.99 * img[idx].g());
			data[idx * 3 + 2] = char(255.99 * img[idx].r());
		}
	}
	stop = clock();
	timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
	std::cerr << "CPU took " << timer_seconds << " seconds.\n\n";

	writeBMP("imgCPU-prueba.bmp", data, w, h);
	printf("Imagen CPU creada.\n");

	std::cerr << "--- GPU ---\n";
	start = clock();
	rayTracingGPU(img, w, h, ns);

	for (int i = h - 1; i >= 0; i--) {
		for (int j = 0; j < w; j++) {
			size_t idx = i * w + j;
			data[idx * 3 + 0] = char(255.99 * img[idx].b());
			data[idx * 3 + 1] = char(255.99 * img[idx].g());
			data[idx * 3 + 2] = char(255.99 * img[idx].r());
		}
	}
	stop = clock();
	timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
	std::cerr << "GPU took " << timer_seconds << " seconds.\n";

	writeBMP("imgGPU-prueba.bmp", data, w, h);
	printf("Imagen GPU creada.\n");

	free(data);
	hipDeviceReset();

	getchar();
	return (0);
}
